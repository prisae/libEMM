/* CSEM FDTD modeling using Nvidia GPU
 *--------------------------------------------------------------------
 *
 *   Copyright (c) 2020, Harbin Institute of Technology, China
 *   Author: Pengliang Yang
 *   E-mail: ypl.2100@gmail.com
 *   Homepage: https://yangpl.wordpress.com
 *--------------------------------------------------------------------*/
#include <mpi.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>


extern "C"{
#include "cstd.h"
#include "acqui.h"
#include "emf.h"
#include "interp.h"
#include "constants.h"
}

#include "cuda_fdtd.cuh"


float c11, c21, c12, c22, c13, c23;
float *d_v3, *d_v3s;

hipError_t status;
dim3 dimBlock, dimGrid, dimGrid_dtft;

float *d_inveps11, *d_inveps22, *d_inveps33;
float *d_E1, *d_E2, *d_E3, *d_H1, *d_H2, *d_H3;
float *d_curlE1, *d_curlE2, *d_curlE3, *d_curlH1, *d_curlH2, *d_curlH3;
float *d_memD2E1, *d_memD3E1, *d_memD1E2, *d_memD3E2, *d_memD1E3, *d_memD2E3;
float *d_memD2H1, *d_memD3H1, *d_memD1H2, *d_memD3H2, *d_memD1H3, *d_memD2H3;
float *d_a1, *d_b1, *d_a2, *d_b2, *d_a3, *d_b3;
float *d_omegas;

hipFloatComplex *d_fwd_E1, *d_fwd_E2, *d_fwd_E3;
hipFloatComplex *d_fwd_H1, *d_fwd_H2, *d_fwd_H3;
hipFloatComplex *d_backup, *d_expfactor;
int *d_corner_id, *h_ncorner;

hipfftHandle fftPlan;
hipFloatComplex *d_sH1kxky, *d_sH2kxky, *d_emfft, *d_emfft0;
float *d_sE12kxky;


int *d_rg_src_i1, *d_rg_src_i2, *d_rg_src_i3;
float *d_rg_src_w1, *d_rg_src_w2, *d_rg_src_w3;
int *d_sg_src_i1, *d_sg_src_i2, *d_sg_src_i3;
float *d_sg_src_w1, *d_sg_src_w2, *d_sg_src_w3;

int *d_chsrc;



void cuda_fdtd_init(emf_t *emf)
{
  int ic, *h_chsrc;
  int i1, i2, i3;

  int nchsrc = emf->nchsrc;
  int corner_id[8];

  //4-th order staggered FD, backward difference using shared memory:
  //c1*(D[0]-D[-1])+c2*(D[1]-D[-2])
  c11 = fd_c1/emf->d1;
  c21 = fd_c2/emf->d1;
  c12 = fd_c1/emf->d2;
  c22 = fd_c2/emf->d2;
  c13 = fd_c1/emf->d3;
  c23 = fd_c2/emf->d3;

  /* allocate memory on device */
  hipMalloc(&d_inveps11, emf->n123pad*sizeof(float));
  hipMalloc(&d_inveps22, emf->n123pad*sizeof(float));
  hipMalloc(&d_inveps33, emf->n123pad*sizeof(float));
  hipMalloc(&d_E1, emf->n123pad*sizeof(float));
  hipMalloc(&d_E2, emf->n123pad*sizeof(float));
  hipMalloc(&d_E3, emf->n123pad*sizeof(float));
  hipMalloc(&d_H1, emf->n123pad*sizeof(float));
  hipMalloc(&d_H2, emf->n123pad*sizeof(float));
  hipMalloc(&d_H3, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE1, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE2, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlE3, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH1, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH2, emf->n123pad*sizeof(float));
  hipMalloc(&d_curlH3, emf->n123pad*sizeof(float));
  hipMalloc(&d_memD2E1, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3E1, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1E2, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3E2, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1E3, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2E3, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2H1, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3H1, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1H2, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD3H2, emf->n1pad*emf->n2pad*2*emf->nb*sizeof(float));
  hipMalloc(&d_memD1H3, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMalloc(&d_memD2H3, emf->n1pad*2*emf->nb*emf->n3pad*sizeof(float));
  hipMalloc(&d_fwd_E1, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_E2, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_E3, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_H1, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_H2, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_fwd_H3, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMalloc(&d_a1, emf->nb*sizeof(float));
  hipMalloc(&d_b1, emf->nb*sizeof(float));
  hipMalloc(&d_a2, emf->nb*sizeof(float));
  hipMalloc(&d_b2, emf->nb*sizeof(float));
  hipMalloc(&d_a3, emf->nb*sizeof(float));
  hipMalloc(&d_b3, emf->nb*sizeof(float));
  hipMalloc(&d_v3, 2*emf->rd*emf->n3pad*sizeof(float));
  hipMalloc(&d_v3s, 2*emf->rd*emf->n3pad*sizeof(float));


  hipMalloc(&d_corner_id, 8*sizeof(int));
  hipHostAlloc(&h_ncorner, sizeof(int), hipHostMallocMapped);	
  hipMalloc(&d_backup, 8*sizeof(hipFloatComplex));
  hipMalloc(&d_expfactor, emf->nfreq*emf->nt*sizeof(hipFloatComplex));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - fdtd !\n"); exit(0); }
  
  //initialize memory on device
  hipMemcpy(d_inveps11, emf->inveps11[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_inveps22, emf->inveps22[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_inveps33, emf->inveps33[0][0], emf->n123pad*sizeof(float), hipMemcpyHostToDevice);
  hipMemset(d_E1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_E2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_E3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_H3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlE3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH1, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH2, 0, emf->n123pad*sizeof(float));
  hipMemset(d_curlH3, 0, emf->n123pad*sizeof(float));
  hipMemset(d_memD2E1, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3E1, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1E2, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3E2, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1E3, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2E3, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2H1, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3H1, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1H2, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD3H2, 0, 2*emf->nb*emf->n1pad*emf->n2pad*sizeof(float));
  hipMemset(d_memD1H3, 0, 2*emf->nb*emf->n2pad*emf->n3pad*sizeof(float));
  hipMemset(d_memD2H3, 0, 2*emf->nb*emf->n1pad*emf->n3pad*sizeof(float));
  hipMemset(d_fwd_E1, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_E2, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_E3, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_H1, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_H2, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemset(d_fwd_H3, 0, emf->n123pad*emf->nfreq*sizeof(hipFloatComplex));
  hipMemcpy(d_a1, emf->a1, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b1, emf->b1, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a2, emf->a2, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b2, emf->b2, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_a3, emf->a3, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b3, emf->b3, emf->nb*sizeof(float), hipMemcpyHostToDevice);
  if(emf->nugrid){
    hipMemcpy(d_v3, emf->v3[0], 2*emf->rd*emf->n3pad*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v3s, emf->v3s[0], 2*emf->rd*emf->n3pad*sizeof(float), hipMemcpyHostToDevice);
  }


  i1 = emf->nbe;
  i2 = emf->nbe;
  i3 = emf->nbe;
  corner_id[0] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe;
  i3 = emf->nbe;
  corner_id[1] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe;
  corner_id[2] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[3] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 -1;
  i2 = emf->nbe + emf->n2 -1;
  i3 = emf->nbe;
  corner_id[4] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[5] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[6] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  i1 = emf->nbe + emf->n1 - 1;
  i2 = emf->nbe + emf->n2 - 1;
  i3 = emf->nbe + emf->n3 - 1;
  corner_id[7] = i1 + emf->n1pad*(i2 + emf->n2pad*i3);
  hipMemcpy(d_corner_id, corner_id, 8*sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_backup, 0, 8*sizeof(hipFloatComplex));
  hipMemcpy(d_expfactor, &emf->expfactor[0][0], emf->nfreq*emf->nt*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    
  h_chsrc = alloc1int(nchsrc);
  for(ic=0; ic<nchsrc; ++ic) {
    if     (strcmp(emf->chsrc[ic],"Ex")==0) h_chsrc[ic] = 1;
    else if(strcmp(emf->chsrc[ic],"Ey")==0) h_chsrc[ic] = 2;
    else if(strcmp(emf->chsrc[ic],"Ez")==0) h_chsrc[ic] = 3;
    else if(strcmp(emf->chsrc[ic],"Hx")==0) h_chsrc[ic] = 4;
    else if(strcmp(emf->chsrc[ic],"Hy")==0) h_chsrc[ic] = 5;
    else if(strcmp(emf->chsrc[ic],"Hz")==0) h_chsrc[ic] = 6;
  }
  hipMalloc(&d_chsrc, nchsrc*sizeof(int));
  hipMemcpy(d_chsrc, h_chsrc, nchsrc*sizeof(int), hipMemcpyHostToDevice);
  free(h_chsrc);

  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to initialize memory on device - fdtd !\n"); exit(0); }

}

void cuda_fdtd_close()
{
  /* free memory on device */
  hipFree(d_inveps11);
  hipFree(d_inveps22);
  hipFree(d_inveps33);
  hipFree(d_E1);
  hipFree(d_E2);
  hipFree(d_E3);
  hipFree(d_H1);
  hipFree(d_H2);
  hipFree(d_H3);
  hipFree(d_curlE1);
  hipFree(d_curlE2);
  hipFree(d_curlE3);
  hipFree(d_curlH1);
  hipFree(d_curlH2);
  hipFree(d_curlH3);
  hipFree(d_memD2E1);
  hipFree(d_memD3E1);
  hipFree(d_memD1E2);
  hipFree(d_memD3E2);
  hipFree(d_memD1E3);
  hipFree(d_memD2E3);
  hipFree(d_memD2H1);
  hipFree(d_memD3H1);
  hipFree(d_memD1H2);
  hipFree(d_memD3H2);
  hipFree(d_memD1H3);
  hipFree(d_memD2H3);
  hipFree(d_fwd_E1);
  hipFree(d_fwd_E2);
  hipFree(d_fwd_E3);
  hipFree(d_fwd_H1);
  hipFree(d_fwd_H2);
  hipFree(d_fwd_H3);
  hipFree(d_a1);
  hipFree(d_b1);
  hipFree(d_a2);
  hipFree(d_b2);
  hipFree(d_a3);
  hipFree(d_b3);
  hipFree(d_v3);
  hipFree(d_v3s);

  hipFree(d_corner_id);
  hipHostFree(h_ncorner);
  hipFree(d_backup);
  hipFree(d_expfactor);
  
  hipFree(d_chsrc);
}


void cuda_airwave_bc_init(emf_t *emf)
{
  int n1fft = emf->n1fft;
  int n2fft = emf->n2fft;
  
  // create FFT plan
  hipfftPlan2d(&fftPlan, n1fft, n2fft, HIPFFT_C2C);
  hipMalloc(&d_sH1kxky, n1fft*n2fft*emf->rd*sizeof(hipFloatComplex));
  hipMalloc(&d_sH2kxky, n1fft*n2fft*emf->rd*sizeof(hipFloatComplex));
  hipMalloc(&d_sE12kxky, n1fft*n2fft*(emf->rd-1)*sizeof(float));
  hipMalloc(&d_emfft, n1fft*n2fft*sizeof(hipFloatComplex));
  hipMalloc(&d_emfft0, n1fft*n2fft*sizeof(hipFloatComplex));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - airwave !\n"); exit(0); }

  hipMemcpy(d_sH1kxky, &emf->sH1kxky[0][0][0], n1fft*n2fft*emf->rd*sizeof(float _Complex), hipMemcpyHostToDevice);
  hipMemcpy(d_sH2kxky, &emf->sH2kxky[0][0][0], n1fft*n2fft*emf->rd*sizeof(float _Complex), hipMemcpyHostToDevice);
  if(emf->rd>1) hipMemcpy(d_sE12kxky, &emf->sE12kxky[0][0][0], n1fft*n2fft*(emf->rd-1)*sizeof(float), hipMemcpyHostToDevice);
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to initialize memory on device - airwave!\n"); exit(0); }
  
}


void cuda_airwave_bc_close()
{
  hipfftDestroy(fftPlan);
  hipFree(d_sH1kxky);
  hipFree(d_sH2kxky);
  hipFree(d_sE12kxky);
  hipFree(d_emfft);
  hipFree(d_emfft0);
}



void cuda_interpolation_init(acqui_t *acqui, emf_t *emf, interp_t *interp_rg, interp_t *interp_sg)
{
  int rd = emf->rd;

  hipMalloc(&d_rg_src_i1, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_i2, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_i3, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_rg_src_w1, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_rg_src_w2, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_rg_src_w3, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));

  hipMalloc(&d_sg_src_i1, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_i2, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_i3, acqui->nsrc*acqui->nsubsrc*sizeof(int));
  hipMalloc(&d_sg_src_w1, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_sg_src_w2, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  hipMalloc(&d_sg_src_w3, 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float));
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to allocate memory on device - interpolation!\n"); exit(0);  }

  //-------------------------------------------------------------------------
  hipMemcpy(d_rg_src_i1, interp_rg->src_i1[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_i2, interp_rg->src_i2[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_i3, interp_rg->src_i3[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w1, interp_rg->src_w1[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w2, interp_rg->src_w2[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_rg_src_w3, interp_rg->src_w3[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_sg_src_i1, interp_sg->src_i1[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_i2, interp_sg->src_i2[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_i3, interp_sg->src_i3[0], acqui->nsrc*acqui->nsubsrc*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w1, interp_sg->src_w1[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w2, interp_sg->src_w2[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_sg_src_w3, interp_sg->src_w3[0][0], 2*rd*acqui->nsrc*acqui->nsubsrc*sizeof(float), hipMemcpyHostToDevice);

  status = hipGetLastError();
  if(hipSuccess!=status) { printf("Failed to initialize memory on device - interpolation !\n"); exit(0); }
}

void cuda_interpolation_close()
{
  hipFree(d_rg_src_i1);
  hipFree(d_rg_src_i2);
  hipFree(d_rg_src_i3);
  hipFree(d_rg_src_w1);
  hipFree(d_rg_src_w2);
  hipFree(d_rg_src_w3);

  hipFree(d_sg_src_i1);
  hipFree(d_sg_src_i2);
  hipFree(d_sg_src_i3);
  hipFree(d_sg_src_w1);
  hipFree(d_sg_src_w2);
  hipFree(d_sg_src_w3);
}


extern "C"
void cuda_modeling(acqui_t *acqui, emf_t *emf, interp_t *interp_rg, interp_t *interp_sg)
{  
  hipEvent_t start, stop;
  int it, ic;
  double t_updateE,t_updateH,t_injectE,t_injectH,t_curlE, t_curlH,t_dtft,t_conv, t_airwave, t0;
  float mstimer;
  float normalization = 1./(emf->n1fft*emf->n2fft);
  
  hipSetDevice(0);// initialize device, default device=0;
  status = hipGetLastError();
  if (hipSuccess!=status) { printf("Failed to initialize device!\n"); exit(0); }

  /*==========================================================*/
  cuda_interpolation_init(acqui, emf, interp_rg, interp_sg);
  cuda_fdtd_init(emf);
  if(emf->airwave) cuda_airwave_bc_init(emf);
    
  dimBlock.x = BlockSize1; 
  dimBlock.y = BlockSize2;
  dimGrid.x = (emf->n1pad+BlockSize1-1)/BlockSize1;
  dimGrid.y = (emf->n2pad+BlockSize2-1)/BlockSize2;
  dimGrid_dtft.x = (emf->n1fft+BlockSize1-1)/BlockSize1;
  dimGrid_dtft.y = (emf->n2fft+BlockSize2-1)/BlockSize2;
  if(emf->verb){
    printf("dimBlock.x=%d \n", dimBlock.x);
    printf("dimBlock.y=%d \n", dimBlock.y);
    printf("dimGrid.x=%d \n", dimGrid.x);
    printf("dimGrid.y=%d \n", dimGrid.y);
    printf("dimGrid_dtft.x=%d \n", dimGrid_dtft.x);
    printf("dimGrid_dtft.y=%d \n", dimGrid_dtft.y);

    t0 = 0;
    t_curlE = 0.;
    t_injectH = 0.;
    t_updateH = 0.;
    t_curlH = 0.;
    t_injectE = 0.;
    t_updateE = 0.;
    t_airwave = 0;
    t_dtft= 0.;
    t_conv=0.;
  }

  hipEventCreate(&start);	
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(it=0; it<emf->nt; it++){
    if(emf->verb && it%50==0) printf("it---- %d\n", it);

    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    if(emf->nugrid)
      cuda_fdtd_curlE_nugrid<<<dimGrid,dimBlock>>>
	(d_E1, d_E2, d_E3, d_curlE1, d_curlE2, d_curlE3, d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
	 d_memD1E2, d_memD1E3, d_memD2E1, d_memD2E3, d_memD3E1, d_memD3E2, 
	 c11, c21, c12, c22, d_v3,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave);
    else
      cuda_fdtd_curlE<<<dimGrid,dimBlock>>>
	(d_E1, d_E2, d_E3, d_curlE1, d_curlE2, d_curlE3, d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
	 d_memD1E2, d_memD1E3, d_memD2E1, d_memD2E3, d_memD3E1, d_memD3E2, 
	 c11, c21, c12, c22, c13, c23,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute curlE on device!\n"); exit(0); }    
    if(emf->verb) t_curlE += MPI_Wtime()-t0;

    
    if(emf->verb) t0 = MPI_Wtime();
    cuda_inject_magnetic_source<<<(emf->nchsrc*acqui->nsrc*acqui->nsubsrc+BlockSize-1)/BlockSize,BlockSize>>>
      (d_rg_src_i1, d_rg_src_i2, d_rg_src_i3, d_rg_src_w1, d_rg_src_w2, d_rg_src_w3,
       d_sg_src_i1, d_sg_src_i2, d_sg_src_i3, d_sg_src_w1, d_sg_src_w2, d_sg_src_w3,
       d_curlE1, d_curlE2, d_curlE3, d_chsrc,
       emf->stf[it], emf->d1, emf->d2, emf->d3, emf->nchsrc, acqui->nsrc, acqui->nsubsrc,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->nbe, emf->rd);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to inject magnetic source on device!\n"); exit(0); }
    if(emf->verb) t_injectH += MPI_Wtime()-t0;

    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_update_H<<<dimGrid,dimBlock>>>(d_H1, d_H2, d_H3, d_curlE1, d_curlE2, d_curlE3,
					     emf->dt, emf->n1pad, emf->n2pad, emf->n3pad);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to update H on device!\n"); exit(0); }    
    if(emf->verb) t_updateH += MPI_Wtime()-t0;


    if(emf->verb) t0 = MPI_Wtime();
    if(emf->airwave){
      cuda_airwave_bc_copy<<<dimGrid_dtft,dimBlock>>>
    	(d_emfft, &d_H3[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft0, HIPFFT_FORWARD);//FFT into wavenumber domain

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dtft,dimBlock>>>(d_emfft, d_sH1kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dtft,dimBlock>>>
	(&d_H1[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dtft,dimBlock>>>(d_emfft, &d_sH1kxky[emf->n1fft*emf->n2fft], emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dtft,dimBlock>>>
	(&d_H1[emf->n1pad*emf->n2pad*(emf->nbe-2)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dtft,dimBlock>>>(d_emfft, d_sH2kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dtft,dimBlock>>>
	(&d_H2[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      hipMemcpy(d_emfft, d_emfft0, emf->n1fft*emf->n2fft*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
      cuda_airwave_bc_scale_FH<<<dimGrid_dtft,dimBlock>>>(d_emfft, &d_sH2kxky[emf->n1fft*emf->n2fft], emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid_dtft,dimBlock>>>
	(&d_H2[emf->n1pad*emf->n2pad*(emf->nbe-2)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      status = hipGetLastError();
      if (hipSuccess!=status) { printf("Failed to handle air-water interface!\n"); exit(0); }
    }
    if(emf->verb) t_airwave += MPI_Wtime()-t0;
    
    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    if(emf->nugrid)
      cuda_fdtd_curlH_nugrid<<<dimGrid,dimBlock>>>
	(d_H1, d_H2, d_H3, d_curlH1, d_curlH2, d_curlH3,
	 d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
	 d_memD1H2, d_memD1H3, d_memD2H1,
	 d_memD2H3, d_memD3H1, d_memD3H2, 
	 d_inveps11, d_inveps22, d_inveps33,
	 c11, c21, c12, c22, d_v3s,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave);
    else
      cuda_fdtd_curlH<<<dimGrid,dimBlock>>>
	(d_H1, d_H2, d_H3, d_curlH1, d_curlH2, d_curlH3,
	 d_a1, d_b1, d_a2, d_b2, d_a3, d_b3,
	 d_memD1H2, d_memD1H3, d_memD2H1,
	 d_memD2H3, d_memD3H1, d_memD3H2, 
	 d_inveps11, d_inveps22, d_inveps33,
	 c11, c21, c12, c22, c13, c23,
	 emf->n1pad, emf->n2pad, emf->n3pad, emf->nb, emf->nbe, emf->airwave);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute curlH on device!\n"); exit(0); }    
    if(emf->verb) t_curlH += MPI_Wtime()-t0;

    if(emf->verb) t0 = MPI_Wtime();
    cuda_inject_electric_source<<<(emf->nchsrc*acqui->nsrc*acqui->nsubsrc+BlockSize-1)/BlockSize,BlockSize>>>
      (d_rg_src_i1, d_rg_src_i2, d_rg_src_i3, d_rg_src_w1, d_rg_src_w2, d_rg_src_w3,
       d_sg_src_i1, d_sg_src_i2, d_sg_src_i3, d_sg_src_w1, d_sg_src_w2, d_sg_src_w3,
       d_inveps11, d_inveps22, d_inveps33, d_curlH1, d_curlH2, d_curlH3, d_chsrc, 
       emf->stf[it], emf->d1, emf->d2, emf->d3, emf->nchsrc, acqui->nsrc, acqui->nsubsrc,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->nbe, emf->rd);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to inject electric source on device!\n"); exit(0); }
    if(emf->verb) t_injectE += MPI_Wtime()-t0;
    
    if(emf->verb) t0 = MPI_Wtime();
    cuda_fdtd_update_E<<<dimGrid,dimBlock>>>
      (d_E1, d_E2, d_E3, d_curlH1, d_curlH2, d_curlH3, d_inveps11, d_inveps22, d_inveps33,
       emf->n1pad, emf->n2pad, emf->n3pad, emf->dt);
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to update E on device!\n"); exit(0); }    
    if(emf->verb) t_updateE += MPI_Wtime()-t0;


    if(emf->verb) t0 = MPI_Wtime();
    if(emf->airwave){
      cuda_airwave_bc_copy<<<dimGrid_dtft,dimBlock>>>
	(d_emfft, &d_E1[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_FORWARD);//FFT into wavenumber domain
      cuda_airwave_bc_scale_FE<<<dimGrid_dtft,dimBlock>>>(d_emfft, d_sE12kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid,dimBlock>>>
    	(&d_E1[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      cuda_airwave_bc_copy<<<dimGrid_dtft,dimBlock>>>
	(d_emfft, &d_E2[emf->n1pad*emf->n2pad*emf->nbe], emf->n1pad, emf->n2pad, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_FORWARD);//FFT into wavenumber domain
      cuda_airwave_bc_scale_FE<<<dimGrid_dtft,dimBlock>>>(d_emfft, d_sE12kxky, emf->n1fft, emf->n2fft);
      hipfftExecC2C(fftPlan, d_emfft, d_emfft, HIPFFT_BACKWARD);//IFFT back to space domain
      cuda_airwave_bc_back2emf<<<dimGrid,dimBlock>>>
    	(&d_E2[emf->n1pad*emf->n2pad*(emf->nbe-1)], d_emfft, emf->n1pad, emf->n2pad, emf->n1fft, normalization);

      status = hipGetLastError();
      if (hipSuccess!=status) { printf("Failed to handle air-water interface!\n"); exit(0); }
    }
    if(emf->verb) t_airwave += MPI_Wtime()-t0;
    
    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    for(ic=0; ic<emf->nchrec; ++ic) {
      if     (strcmp(emf->chrec[ic],"Ex")==0) 
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_E1, &d_expfactor[it*emf->nfreq], d_E1, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
      else if(strcmp(emf->chrec[ic],"Ey")==0) 
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_E2, &d_expfactor[it*emf->nfreq], d_E2, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
      else if(strcmp(emf->chrec[ic],"Ez")==0)
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_E3, &d_expfactor[it*emf->nfreq], d_E3, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
      else if(strcmp(emf->chrec[ic],"Hx")==0) 
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_H1, &d_expfactor[it*emf->nfreq], d_H1, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
      else if(strcmp(emf->chrec[ic],"Hy")==0) 
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_H2, &d_expfactor[it*emf->nfreq], d_H2, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
      else if(strcmp(emf->chrec[ic],"Hz")==0) 
	cuda_dtft_emf<<<dimGrid,dimBlock>>>(d_fwd_H3, &d_expfactor[it*emf->nfreq], d_H3, 
					    emf->nb, emf->n123pad, emf->n1pad, emf->n2pad, emf->n3pad, emf->nfreq);
    }
    status = hipGetLastError();
    if (hipSuccess!=status) { printf("Failed to compute DFT of E + H on device!\n"); exit(0); }    
    if(emf->verb) t_dtft += MPI_Wtime()-t0;


    /*--------------------------------------------------------------*/
    if(emf->verb) t0 = MPI_Wtime();
    if(it%100==0){/* convergence check */
      cuda_check_convergence<<<1,8>>>(d_corner_id, d_fwd_E1, d_backup, h_ncorner);
      if(emf->verb) printf("%d corners of the cube converged!\n", h_ncorner[0]);
      if(h_ncorner[0]==8) { printf("converge after %d steps\n", it); break; }/* all 8 corners converged, exit now */
    }
    if(emf->verb) t_conv += MPI_Wtime()-t0;
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&mstimer, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  if(emf->verb) {
    t0 = t_curlH + t_injectE + t_updateE
      + t_curlE + t_injectH + t_updateH
      + t_airwave + t_dtft + t_conv;
    FILE *fp = fopen("time_info.txt", "w");
    fprintf(fp, "curlE   \t %e\n", t_curlE);
    fprintf(fp, "injectH \t %e\n", t_injectH);
    fprintf(fp, "udpateH \t %e\n", t_updateH);
    fprintf(fp, "curlH   \t %e\n", t_curlH);
    fprintf(fp, "injectE \t %e\n", t_injectE);
    fprintf(fp, "udpateE \t %e\n", t_updateE);
    fprintf(fp, "airwave \t %e\n", t_airwave);
    fprintf(fp, "dtft    \t %e\n", t_dtft);
    fprintf(fp, "conv    \t %e\n", t_conv);
    fprintf(fp, "total   \t %e\n", mstimer*1e-3);    
    fclose(fp);
    
    printf("-------------- elapsed time --------------------\n");
    printf(" compute curlE:           %e s\n", t_curlE);
    printf(" inject magnetic source:  %e s\n", t_injectH);
    printf(" update magnetic field:   %e s\n", t_updateH);

    printf(" compute curlH:           %e s\n", t_curlH);
    printf(" inject electric source:  %e s\n", t_injectE);
    printf(" update electric field:   %e s\n", t_updateE);

    printf(" Airwave computation:     %e s\n", t_airwave);
    printf(" DTFT EM field:           %e s\n", t_dtft);
    printf(" convergence check:       %e s\n", t_conv);
    printf(" Total modeling time:     %e s\n", mstimer*1.e-3);
    printf("------------------------------------------------\n");
  }
  hipMemcpy(&emf->fwd_E1[0][0][0][0], &d_fwd_E1[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&emf->fwd_E2[0][0][0][0], &d_fwd_E2[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&emf->fwd_E3[0][0][0][0], &d_fwd_E3[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&emf->fwd_H1[0][0][0][0], &d_fwd_H1[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&emf->fwd_H2[0][0][0][0], &d_fwd_H2[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&emf->fwd_H3[0][0][0][0], &d_fwd_H3[0], emf->nfreq*emf->n123pad*sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

  cuda_interpolation_close();
  cuda_fdtd_close(); 
  if(emf->airwave) cuda_airwave_bc_close();

}
